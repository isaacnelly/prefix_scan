#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

__global__ void kogge_stone_inclusive(unsigned int* output, unsigned int* input){
    // boilerplate
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = input[i];
    __synchthreads();

    for (unsigned int stride = 1; stride <= BLOCK_DIM/2; stride*=2){

    }
}

int main(){

    return 0;
}