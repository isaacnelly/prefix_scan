#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256

/**
 * @brief kogge stone algorithm, inclusive w/ parallelization
 * 
 * @param output 
 * @param input 
 * @param partialSums 
 * @return __global__ 
 */
__global__ void kogge_stone_inclusive(unsigned int* output, unsigned int* input, 
    unsigned int* partialSums){
    // boilerplate
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = input[i];
    __synchthreads();

    for (unsigned int stride = 1; stride <= BLOCK_DIM/2; stride*=2){
        float v;
        if(threadIdx.x >= stride){
            v = output[i - stride];
        }
        __synchthreads();
        if(threadIdx.x >= stride){
            output[i] += v;
        }
        __synchthreads();
    }

    if(threadIdx.x == BLOCK_DIM - 1){
        partialSums[blockIdx.x] = output[i];
    }
}

/**
 * @brief generates a random array, length provided in main. 
 * 
 * @param length 
 * @return __global 
 */
__global void generateRandomArray(unsigned int* array, unsigned int length, unsigned long long seed){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < length){
        hiprandState state;
        hiprand_init(seed,i,0,&state);
        array[i] = hiprand(&state);
    }
}

int main(){
    int n = 1024;
    unsigned int* RandIntArray_d;
    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    hipMalloc((void**)&RandIntArray, n*sizeof(unsigned int));

    generateRandomArray<<<blocks,THREADS_PER_BLOCK>>>(RandIntArray,n,0);


    return 0;
}