#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

__global__ void kogge_stone_inclusive(unsigned int* output, unsigned int* input, 
    unsigned int* partialSums){
    // boilerplate
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = input[i];
    __synchthreads();

    for (unsigned int stride = 1; stride <= BLOCK_DIM/2; stride*=2){
        float v;
        if(threadIdx.x >= stride){
            v = output[i - stride];
        }
        __synchthreads();
        if(threadIdx.x >= stride){
            output[i] += v;
        }
        __synchthreads();
    }

    if(threadIdx.x == BLOCK_DIM - 1){
        partialSums[blockIdx.x] = output[i];
    }
}


int main(){



    return 0;
}